#include "hip/hip_runtime.h"
//
// Created by lepet on 4/6/2022.
//

#include <thrust/find.h>
#include <thrust/zip_function.h>
#include <thrust/binary_search.h>

#include "states.h"

using Node = uint64_t;
using Value = uint8_t;
enum Edge : uint8_t {
    NORTH = 0,
    EAST = 1,
    SOUTH = 2,
    WEST = 3,
};

using States = empire::States<Node, Edge, Value>;
using HostStates = empire::HostStates<Node, Edge, Value>;

struct Expansion {
    Node target;

    explicit Expansion(Node target) : target(target) {}

    // Heuristic
    [[nodiscard]]
    __host__ __device__
    Value heuristic(Node node) const {
        Node mask = 0xf;
        Value result = 0;
        for (int i = 0; i < 16; ++i, mask <<= 4) {
            auto x = (mask & node) >> (4 * i);
            auto y = (mask & target) >> (4 * i);
            if (x != y && x != 0) ++result;
        }
        return result;
    }

    // Expansion
    __host__ __device__
    void operator()(
            const Node& node,
            const Value& step,
            const size_t& direction,
            Node& out_node,
            Value& out_step,
            Value& out_score,
            Edge& out_parent
    ) const {
        Node mask = 0xf;
        int x = -1, y = -1;
        for (int i = 0; i < 16; ++i, mask <<= 4) {
            if ((node & mask) == 0) {
                x = i / 4;
                y = i % 4;
                break;
            }
        }

        if (direction == NORTH && x > 0) {
            auto selected = node & (mask >> 16);
            out_node = (node | (selected << 16)) ^ selected;
            out_step = step + 1;
            out_score = step + heuristic(node);
            out_parent = SOUTH;
            return;
        }

        if (direction == EAST && y < 3) {
            auto selected = node & (mask << 4);
            out_node = (node | (selected >> 4)) ^ selected;
            out_step = step + 1;
            out_score = step + heuristic(node);
            out_parent = WEST;
            return;
        }

        if (direction == SOUTH && x < 3) {
            auto selected = node & (mask << 16);
            out_node = (node | (selected >> 16)) ^ selected;
            out_step = step + 1;
            out_score = step + heuristic(node);
            out_parent = NORTH;
            return;
        }

        if (direction == WEST && y > 0) {
            auto selected = node & (mask >> 4);
            out_node = (node | (selected << 4)) ^ selected;
            out_step = step + 1;
            out_score = step + heuristic(node);
            out_parent = EAST;
            return;
        }

        out_node = 0;
    }
};

struct NodeComp {
    __host__ __device__
    bool operator()(const Node& lhs, const Node& rhs) {
        if (lhs == 0) return false;
        if (rhs == 0) return true;
        return lhs < rhs;
    }
};

struct StateReduce {
    template<typename Tuple>
    __host__ __device__
    Tuple operator()(const Tuple& lhs, const Tuple& rhs) {
        return thrust::get<1>(lhs) < thrust::get<1>(rhs) ? lhs : rhs;
    }
};

struct StateSelect {
    __host__ __device__
    void operator()(
            const Node& control_node,
            const Value& control_score,
            const Value& test_score,
            Node& out_node
    ) {
        if (control_node == out_node && control_score <= test_score)
            out_node = 0;
    }
};

struct StatePartition {
    template<typename Tuple>
    __host__ __device__
    bool operator()(const Tuple& tuple) {
        return thrust::get<0>(tuple) != 0;
    }
};

auto make_expand_iter(const States& input, States& output, size_t stride, size_t x = 0) {
    using namespace thrust::placeholders;
    auto expand_counter = thrust::make_counting_iterator(x);
    auto stride_counter = thrust::make_transform_iterator(expand_counter, _1 % stride);
    auto direction_iter = thrust::make_transform_iterator(expand_counter, _1 / stride);

    auto input_nodes_iter = thrust::make_permutation_iterator(input.nodes.begin(), stride_counter);
    auto input_steps_iter = thrust::make_permutation_iterator(input.steps.begin(), stride_counter);

    return thrust::make_zip_iterator(
            input_nodes_iter,
            input_steps_iter,
            direction_iter,
            output.nodes.begin() + x,
            output.steps.begin() + x,
            output.scores.begin() + x,
            output.parents.begin() + x
    );
}

auto make_selection_iter(
        const thrust::device_vector<Node>& indices,
        const States& close,
        States& dedup,
        size_t x = 0
) {
    auto indices_iter = indices.begin() + x;
    return thrust::make_zip_iterator(
            thrust::make_permutation_iterator(close.keys(), indices_iter),
            thrust::make_permutation_iterator(close.keys_score(), indices_iter),
            dedup.keys_score(x),
            dedup.keys(x)
    );
}

void print_node(Node node) {
    Node mask = 0xf;
    for (auto i = 0u; i < 4; ++i) {
        for (auto j = 0u; j < 4; ++j) {
            auto n = (mask & node) >> ((4 * i + j) * 4);
            std::cout << n << '\t';
            mask <<= 4;
        }
        std::cout << '\n';
    }
    std::cout << std::endl;
}

int main() {
    Node start = 0xFEDCBA9876543210;
    Node target = 0xAECDF941B8527306;
    Expansion expansion(target);

    States open, close, merge, expand, dedup;
    thrust::device_vector<Node> indices;
    size_t expand_stride = 1024;

    open.reserve(4096);
    close.reserve(4096);
    merge.reserve(4096);

    expand.resize(expand_stride << 2);
    dedup.reserve(expand_stride << 2);
    indices.reserve(expand_stride << 2);

    open.push_back(thrust::make_tuple(start, 0, expansion.heuristic(start), NORTH));
    close = open;

    int iterations = 0;
    for (int i = 0;; ++i) {
        auto stride = std::min(expand_stride, open.size());

        {
            auto expand_size = stride * 4;

            thrust::for_each(
                    make_expand_iter(open, expand, stride),
                    make_expand_iter(open, expand, stride, expand_size),
                    thrust::make_zip_function(expansion)
            );

            // Sort the expanded list by node.
            thrust::sort_by_key(
                    expand.keys(),
                    expand.keys(expand_size),
                    expand.values(),
                    NodeComp()
            );
            // Reduce, first pass.
            dedup.resize(expand_size);
            thrust::reduce_by_key(
                    expand.keys(),
                    expand.keys(expand_size),
                    expand.values(),
                    dedup.keys(),
                    dedup.values(),
                    thrust::equal_to<Node>(),
                    StateReduce()
            );
            auto expand_end = thrust::find(dedup.keys(), dedup.keys(expand_size), 0);
            dedup.resize(expand_end - dedup.keys());
        }

        {
            // Search in close list
            indices.resize(dedup.size());
            thrust::lower_bound(
                    close.keys(),
                    close.keys(close.size()),
                    dedup.keys(),
                    dedup.keys(dedup.size()),
                    indices.begin()
            );

            // Exclude suboptimal states
            thrust::for_each(
                    make_selection_iter(indices, close, dedup),
                    make_selection_iter(indices, close, dedup, dedup.size()),
                    thrust::make_zip_function(StateSelect())
            );
            auto end = thrust::stable_partition(dedup.iter(), dedup.iter(dedup.size()), StatePartition());
            dedup.resize(end - dedup.iter());
        }

        {
            // Update close list.
            // Close list is assumed to be sorted by node.
            merge.resize(close.size() + dedup.size());
            thrust::merge_by_key(
                    close.keys(),
                    close.keys(close.size()),
                    dedup.keys(),
                    dedup.keys(dedup.size()),
                    close.values(),
                    dedup.values(),
                    merge.keys(),
                    merge.values()
            );
            close.resize(merge.size());
            auto ends = thrust::reduce_by_key(
                    merge.keys(),
                    merge.keys(merge.size()),
                    merge.values(),
                    close.keys(),
                    close.values(),
                    thrust::equal_to<Node>(),
                    StateReduce()
            );
            close.resize(ends.first - close.keys());
        }

        {
            // The open list is assumed to be sorted by score.
            // Sort the expanded list by score and merge with open list.
            thrust::sort_by_key(
                    dedup.keys_score(),
                    dedup.keys_score(dedup.size()),
                    dedup.values_score()
            );

            merge.resize(open.size() - stride + dedup.size());
            thrust::merge_by_key(
                    open.keys_score(stride),
                    open.keys_score(open.size()),
                    dedup.keys_score(),
                    dedup.keys_score(dedup.size()),
                    open.values_score(stride),
                    dedup.values_score(),
                    merge.keys_score(),
                    merge.values_score()
            );
            thrust::swap(open, merge);
        }

        if (thrust::binary_search(close.keys(), close.keys(open.size()), target)) {
            iterations = i;
            break;
        }
    }

    HostStates host_open, host_close;
    host_open.copy_from(open);
    host_close.copy_from(close);

    Node node = target;
    while (true) {
        auto iter = std::lower_bound(host_close.nodes.begin(), host_close.nodes.end(), node);
        auto pos = iter - host_close.nodes.begin();
        auto parent = host_close.parents[pos];
        auto step = host_close.steps[pos];
        auto score = host_close.scores[pos];

        std::cout << (int) step << ' ' << (int) score << '\n';
        print_node(node);

        if (node == start) break;

        Node mask = 0xf;
        int x = -1, y = -1;
        for (int i = 0; i < 16; ++i, mask <<= 4) {
            if ((node & mask) == 0) {
                x = i / 4;
                y = i % 4;
                break;
            }
        }
        if (parent == NORTH && x > 0) {
            auto selected = node & (mask >> 16);
            node = (node | (selected << 16)) ^ selected;
        }
        if (parent == EAST && y < 3) {
            auto selected = node & (mask << 4);
            node = (node | (selected >> 4)) ^ selected;
        }
        if (parent == SOUTH && x < 3) {
            auto selected = node & (mask << 16);
            node = (node | (selected >> 16)) ^ selected;
        }
        if (parent == WEST && y > 0) {
            auto selected = node & (mask >> 4);
            node = (node | (selected << 4)) ^ selected;
        }
    }

    std::cout << "Iterations: " << iterations << std::endl;

    return 0;
}